#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <ctime>
#include <cstdlib>

// 
//  Programmer: David Sowsy
//      
//  Game Description: 
//    The game of Nim is a mathematical strategy game where 2 players take turns removing sticks
//    from a single pile, with the goal of forcing the opponent to take the last item.
//    The player who takes the last stick is the winner. The other player is the loser.
// 
//  Notes: 
//    This code maps the kernel to 2 GPUs if more than 1 GPU is present.
//    If only 1 GPU is present, each player is run on a separate stream.
//
//  Revision History: 
//      Initial ideation                          10-02-2023 
//      Initial coding                            10-04-2023
//      Consideration of GPU to stream fallback   10-05-2023
//      Additional output & turn enforcement      10-05-2023

// Helper functions for CUDA randomization
__device__ int GenerateRandomNumber(int max_value, hiprandState_t* state) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  int move = hiprand(&state[id]) % max_value + 1;  // Random move
  return move;
}

__global__ void InitializeCurand(hiprandState_t* state) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(1234 + id, id, 0, &state[id]);
}

// First player
__global__ void Player1(int* pile, hiprandState_t* state, int* sticks_taken) {
  int move = GenerateRandomNumber(*pile, state);
  if (*pile > 0) {
    *pile -= move;
    *sticks_taken = move;
  }
}

// Second player
__global__ void Player2(int* pile, hiprandState_t* state, int* sticks_taken) {
  int move = GenerateRandomNumber(*pile, state);
  if (*pile > 0) {
    *pile -= move;
    *sticks_taken = move; 
  }
}

int main() {
  srand(time(0));  // Seed for random number generation

  int num_gpus = 0;
  hipGetDeviceCount(&num_gpus);  // Get the number of available GPUs

  // Setup game state with a random pile size greater than 10
  int pile_size = rand() % 90 + 10;  // Random number between 10 and 100
  int *d_pile1, *d_pile2;  // Device pointers

  // Track the number of sticks taken back from the kernel.
  int *d_sticks_taken1, *d_sticks_taken2;
  hipMalloc(&d_sticks_taken1, sizeof(int));
  hipMalloc(&d_sticks_taken2, sizeof(int));

  // Allocate memory on the GPU
  hipSetDevice(0);  // Set the first GPU as the current device
  hipMalloc(&d_pile1, sizeof(int));
  hipMemcpy(d_pile1, &pile_size, sizeof(int), hipMemcpyHostToDevice);

  hiprandState_t* d_state1;
  hipMalloc(&d_state1, sizeof(hiprandState_t));
  InitializeCurand<<<1, 1>>>(d_state1);

  hiprandState_t* d_state2;
  hipMalloc(&d_state2, sizeof(hiprandState_t));
  InitializeCurand<<<1, 1>>>(d_state2);
  
  if (num_gpus > 1) {
    hipSetDevice(1);  // Set the second GPU as the current device
    hipMalloc(&d_pile2, sizeof(int));
    hipMemcpy(d_pile2, &pile_size, sizeof(int), hipMemcpyHostToDevice);

    std::cout << "There are " << num_gpus << " present. Executing GPU strategy." << std::endl;
  } else {
    std::cout << "There is only 1 GPU present. Executing streams strategy." << std::endl;
  }

  std::cout << "Initial pile size is " << pile_size << "."<< std::endl;

  int player_turn = 1; 
  // Main game loop
  while (pile_size > 0) {
    // Call player kernels alternatively
    int sticks_taken; 
    if (player_turn == 1) {
      if (num_gpus > 1) {
        Player1<<<1, 1>>>(d_pile1, d_state1, d_sticks_taken1);
        hipDeviceSynchronize();
        hipSetDevice(1);  // Switch to the second GPU for Player 2
      } else {
        hipStream_t stream_player1;
        hipStreamCreate(&stream_player1);
        Player1<<<1, 1, 0, stream_player1>>>(d_pile1, d_state1, d_sticks_taken1);
        hipDeviceSynchronize();
        hipStreamDestroy(stream_player1);
      }
      hipMemcpy(&sticks_taken, d_sticks_taken1, sizeof(int), hipMemcpyDeviceToHost);
      std::cout << "Player 1 takes " << sticks_taken << " stick(s). ";
      player_turn = 2;
    } else {
      if (num_gpus > 1) {
        Player2<<<1, 1>>>(d_pile2, d_state2, d_sticks_taken2);
        hipDeviceSynchronize();
        hipSetDevice(0);  // Switch back to the first GPU for Player 1
      } else {
        hipStream_t stream_player2;
        hipStreamCreate(&stream_player2);
        Player2<<<1, 1, 0, stream_player2>>>(d_pile1, d_state1, d_sticks_taken2);
        hipDeviceSynchronize();
        hipStreamDestroy(stream_player2);
      }
      hipMemcpy(&sticks_taken, d_sticks_taken2, sizeof(int), hipMemcpyDeviceToHost);
      std::cout << "Player 2 takes " << sticks_taken << " stick(s). ";
      player_turn = 1;
    }

    // Copy updated pile_size from GPU to CPU
    hipMemcpy(&pile_size, (num_gpus > 1) ? (pile_size % 2 == 0 ? d_pile1 : d_pile2) : d_pile1, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << pile_size << " stick(s) remain.\n";

    // Check for win condition
    if (pile_size == 0) {
      std::cout << (pile_size % 2 == 0 ? "Player 2" : "Player 1") << " wins!\n";
      break;
    }
  }

  // Free GPU memory
  hipSetDevice(0);
  hipFree(d_pile1);
  hipFree(d_state1);
  if (num_gpus > 1) {
    hipSetDevice(1);
    hipFree(d_pile2);
    hipFree(d_state2);
  }

  hipFree(d_sticks_taken1);
  hipFree(d_sticks_taken2);
  return 0;
}

